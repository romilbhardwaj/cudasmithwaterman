#include "hip/hip_runtime.h"
//A CUDA based implementation of the Smith Waterman Algorithm
//Author: Romil Bhardwaj

#include "hip/hip_runtime.h"
#include ""
#include<time.h>

#include <stdio.h>
#include <stdlib.h>
#define max(a,b) (((a)>(b))?(a):(b))

//Define the costs here
#define indel -1
#define match 2
#define mismatch -1

//CHANGE THIS VALUE TO CHANGE THE NUMBER OF ELEMENTS
const int arraySize = 500;
//CHANGE THIS VALUE TO CHANGE THE NUMBER OF ELEMENTS

hipError_t SWHelper(int (*c)[arraySize+1], const char *a, const char *b, size_t size);


__global__ void SmithWKernelExpand(int (*c)[arraySize+1], const char *a, const char *b, const int *k)		//Declared consts to increase access speed
{
    int i = threadIdx.x+1;
	int j = ((*k)-i)+1;
	int north=c[i][(j)-1]+indel;			//Indel
	int west=c[i-1][j]+indel;
	int northwest;
	if (((int) a[i-1])==((int)b[(j)-1]))
		northwest=c[i-1][(j)-1]+match;		//Match
	else
		northwest=c[i-1][(j)-1]+mismatch;		//Mismatch
    c[i][j] = max(max(north, west),max(northwest,0));
	//c[i][j]=(*k);						//Debugging - Print the antidiag num
}

__global__ void SmithWKernelShrink(int (*c)[arraySize+1], const char *a, const char *b, const int *k)
{
    int i = threadIdx.x+((*k)-arraySize)+1;
	int j = ((*k)-i)+1;
	int north=c[i][(j)-1]+indel;			//Indel
	int west=c[i-1][j]+indel;
	int northwest;
	if (((int) a[i-1])==((int)b[(j)-1]))
		northwest=c[i-1][(j)-1]+match;		//Match
	else
		northwest=c[i-1][(j)-1]+mismatch;		//Mismatch
    c[i][j] = max(max(north, west),max(northwest,0));
	//c[i][j]=(*k);						//Debugging - Print the antidiag num
}

void print(int c[arraySize+1][arraySize+1]){
	int j=0,i=0;
	for (i = 0; i < arraySize+1; i++) {
        for (j = 0; j < arraySize+1; j++) {
            printf("%d \t", c[i][j]);
        }
        printf("\n");
	}
}

void traceback(int c[arraySize+1][arraySize+1], char a[], char b[]){
	int j=0,i=0;
	int maxi=0,maxj=0,max=0;
	for (i = 0; i < arraySize+1; i++) {
        for (j = 0; j < arraySize+1; j++) {
           if(c[i][j]>max){
			   maxi=i;
			   maxj=j;
				max=c[i][j];
		   }
        }
	}
	i=maxi;
	j=maxj;
	printf("The optimal local alignment starts at index %d for a, and index %d for b.\n", i,j);
	while (c[i][j]!=0 && i>=0 && j>=0 ){
		printf("\n");
		if (c[i][j]==c[i-1][(j)-1]+match){		//From match
			i--;
			j--;
			printf("%c -- %c", a[i], b[j]);
		}
		else if (c[i][j]==c[i-1][(j)-1]+mismatch){ //From mismatch
			i--;
			j--;
			printf("%c -- %c", a[i], b[j]);
		}
		else if (c[i][j]==c[i][(j)-1]+indel){	//North
			j--;
			printf("- -- %c", b[j]);
		}
		else{									//Else has to be from West
			i--;
			printf("%c -- -", a[i]);
		}
	}
	
	printf("\n\nThe optimal local alignment ends at index %d for a, and index %d for b.\n", i,j);
}


int main()
{
	char b[arraySize];//{'a','c','a','c','a','c','t','a'};
	char a[arraySize];//{'a','g','c','a','c','a','c','a'};
	
	int i=0;
	
	//Generating the sequences:
	
	srand (time(NULL));
	printf("\nString a is: ");
    for(i=0;i<arraySize;i++)
    {
        int gen1=rand()%4;
        switch(gen1)
        {
            case 0:a[i]='a';
            break;
            case 1: a[i]='c';
            break;
            case 2: a[i]='g';
            break;
            case 3: a[i]='t';
        }
		//a[i]='a';
		printf("%c ", a[i]);
    }

	printf("\nString b is: ");
	for(i=0;i<arraySize;i++)
    {
        int gen1=rand()%4;
        switch(gen1)
        {
            case 0:b[i]='a';
            break;
            case 1: b[i]='c';
            break;
            case 2: b[i]='g';
            break;
            case 3: b[i]='t';
        }
		//b[i]='a';
		printf("%c ", b[i]);
    }
	
	
	printf("\nOkay, generated the string \n");
	int c[arraySize+1][arraySize+1] = { {0} };

	clock_t start=clock();

    // Run the SW Helper function
    hipError_t cudaStatus = SWHelper(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "SWHelper failed!");
        return 1;
    }
	
	clock_t end=clock();

	//Printing the final score matrix. Uncomment this to see the matrix.
	//print(c);

	
	

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	traceback (c,a,b);
	printf("\n\nEnter any number to exit.");
	printf("\n\nTotal time taken is %f seconds\n",(double)(end-start)/CLOCKS_PER_SEC);
	int x;
	scanf("%d", &x);
    return 0;
}

// Helper function for SmithWaterman
hipError_t SWHelper(int (*c)[arraySize+1], const char *a, const char *b, size_t size)
{
    char *dev_a;
    char *dev_b;
	int (*dev_c)[arraySize+1] = {0};
	int (*j)=0;
	int *dev_j;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, (size+1) * (size+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_j, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_j, &j, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, c, (size+1) * (size+1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	int i=0;
	clock_t start1=clock();

    // Launch a kernel on the GPU with one thread for each element.

	//Expanding Phase
	for (i=1; i<size+1; i++){
		cudaStatus = hipMemcpy(dev_j, &i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!", cudaStatus);
			goto Error;
		}
		SmithWKernelExpand<<<1, i>>>(dev_c, dev_a, dev_b, dev_j);
	}

	//Shrink Phase
	for (int k=size-1; k>0; k--, i++){
		cudaStatus = hipMemcpy(dev_j, &i, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		SmithWKernelShrink<<<1, k>>>(dev_c, dev_a, dev_b, dev_j);
	}
	clock_t end1=clock();
    printf("\n\nKernel Time taken is %f seconds\n",(double)(end1-start1)/CLOCKS_PER_SEC);


    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SmithWKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy2D(c,size * size * sizeof(int),dev_c,size * size * sizeof(int),size * size * sizeof(int),size * size * sizeof(int),hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(c, dev_c, (size+1) * (size+1) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
} 
